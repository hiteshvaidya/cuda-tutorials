#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>


#define M 1024  // number of rows in matrix A
#define N 512   // number of columns in matrix B
#define K 1024  // number of columns in matrix A/rows in matrix B
#define BLOCK_SIZE 256 // number of threads per block


void matmul_cpu(float *A, float *B, float *C, int m, int k, int n){
    for (int i = 0; i < m; i++){
        for (int j = 0; j < n; j++){
            float sum = 0.0f;
            for (int l = 0; l < k; l++){
                sum += A[i*k + l] * B[l*n + j];
            }
            C[i*n + j] = sum;
        }
    }
}

__global__ void matmul_kernel(float *A, float *B, float *C, int m, int k, int n){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < n){
        float sum = 0.0f;
        for (int l = 0; l < k; l++){
            sum += A[row * k + l] * B[l * n + col];
        }
        C[row * n + col] = sum;
    }
}

// Initialize matrix with random values
void init_matrix(float *mat, int rows, int cols) {
    for (int i = 0; i < rows * cols; i++) {
        mat[i] = (float)rand() / RAND_MAX;
    }
}

// Function to measure execution time
double get_time() {
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC, &ts);
    return ts.tv_sec + ts.tv_nsec * 1e-9;
}

int main(int argc, char **argv){
    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;
    int size_A = M * K * sizeof(float);
    int size_B = K * N * sizeof(float);
    int size_C = M * N * sizeof(float);

    // Allocate memory on host
    h_A = (float *)malloc(size_A);
    h_B = (float *)malloc(size_B);
    h_C = (float *)malloc(size_C);

    // Allocate memory on device
    hipMalloc(&d_A, size_A);
    hipMalloc(&d_B, size_B);
    hipMalloc(&d_C, size_C);

    // Initialize matrices with random values
    init_matrix(h_A, M, K);
    init_matrix(h_B, K, N);

    // Copy matrices to device
    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

    // Launch kernel
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (M + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // Warm-up runs
    printf("Performing warm-up runs...\n");
    for (int i = 0; i < 3; i++) {
        matmul_cpu(h_A, h_B, h_C, M, K, N);
        matmul_kernel<<<gridDim, blockDim>>>(d_A, d_B, d_C, M, K, N);
        hipDeviceSynchronize();
    }

    // Benchmark CPU implementation
    printf("Benchmarking CPU implementation...\n");
    double cpu_total_time = 0.0;
    for (int i = 0; i < 20; i++) {
        double start_time = get_time();
        matmul_cpu(h_A, h_B, h_C, M, K, N);
        double end_time = get_time();
        cpu_total_time += end_time - start_time;
    }
    double cpu_avg_time = cpu_total_time / 20.0;

    // Benchmark GPU implementation
    printf("Benchmarking GPU implementation...\n");
    double gpu_total_time = 0.0;
    for (int i = 0; i < 20; i++) {
        double start_time = get_time();
        matmul_kernel<<<gridDim, blockDim>>>(d_A, d_B, d_C, M, K, N);
        hipDeviceSynchronize();
        double end_time = get_time();
        gpu_total_time += end_time - start_time;
    }
    double gpu_avg_time = gpu_total_time / 20.0;

    // Print results
    printf("CPU average time: %f microseconds\n", (cpu_avg_time * 1e6f));
    printf("GPU average time: %f microseconds\n", (gpu_avg_time * 1e6f));
    printf("Speedup: %fx\n", cpu_avg_time / gpu_avg_time);

    // Free memory
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
    
}